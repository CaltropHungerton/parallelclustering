﻿
#include "hip/hip_runtime.h"


#include <fstream>
#include <sstream>
#include <iostream>
#include <vector>
#include <random>
#include <string>
#include <ctime>
#include <float.h>
#include <math.h>
#include <tuple>

std::mt19937_64 generator(static_cast<unsigned long>(std::time(0)));
std::uniform_real_distribution<double> distribution(0.0, 10.0);
// quick fisher-yates courtesy of chatgpt
std::vector<int> fisherYatesShuffle(int n, int k) {
    std::vector<int> indices(n); // Array to hold indices [0, 1, ..., n-1]

    // Initialize the array with indices
    for (int i = 0; i < n; ++i) {
        indices[i] = i;
    }

    // Fisher-Yates Shuffle
    for (int i = 0; i < k; ++i) {
        std::uniform_int_distribution<int> distribution(i, n - 1); // Create a uniform distribution
        int j = distribution(generator); // Generate a random index

        // Swap indices[i] and indices[j]
        std::swap(indices[i], indices[j]);
    }

    // Resize the vector to keep only the first k shuffled indices
    indices.resize(k);

    return indices;
}

std::tuple<int, int> getCSVDimensions(const std::string& filePath) 
{
    std::ifstream file(filePath);
    if (!file.is_open()) 
    {
        std::cerr << "Could not open the file: " << filePath << std::endl;
        return {0, 0};
    }

    int rows = 0;
    int cols = 0;
    std::string line;

    while (std::getline(file, line)) 
    {
        rows++;
        std::stringstream lineStream(line);
        std::string cell;
        int tempCols = 0;
        while (std::getline(lineStream, cell, ',')) 
        {
            tempCols++;
        }
        if (rows == 1) 
        {
            cols = tempCols; // Assume all rows have the same number of columns
        }
    }

    return {rows, cols};
}

std::vector<double> readCSVTo1DArray(const std::string& filePath) 
{
    std::ifstream file(filePath);
    if (!file.is_open()) 
    {
        std::cerr << "Could not open the file: " << filePath << std::endl;
        return {};
    }

    std::vector<double> data;
    std::string line;
    while (std::getline(file, line)) 
    {
        std::stringstream lineStream(line);
        std::string cell;
        while (std::getline(lineStream, cell, ',')) 
        {
            data.push_back(std::stod(cell));
        }
    }

    return data;
}

__global__
void update(double* datapoints, double* centroids, int* assigned, int* counts, int num_points, int num_centroids, int dims)
{
    // for now i will only use one block, just use a bunch of threads. I will add to this later.
    int idx = threadIdx.x; // we have num_points threads for this kernel
    double themin = DBL_MAX;
    double dist;
    // now we just iterate striding by dims over the centroid array. then we do euclidean distance between datapoints[idx:idx+dims]
    for (int i = 0; i < num_centroids; i++)
    {
        // euclidean distance
        for (int j = 0; j < dims; j++)
        {
            dist += pow((datapoints[(idx*dims)+j]-centroids[(i*dims)+j]), 2);
        } 
        // sqrt(x) and x are both monotonically increasing, dist is positive, so i don't need to sqrt dist for comparison.
        if (dist < themin)
        {
            themin = dist;
            assigned[idx] = i;
        }
    }
    if (idx < num_centroids*dims)
    {
        centroids[idx] = 0;
    }
    for (int i = 0; i < dims; i++)
    {
        atomicAdd(&centroids[(assigned[idx]*dims)+i], datapoints[(idx*dims)+i]);
    }
    atomicAdd(&counts[assigned[idx]], 1);
    if (idx < num_centroids*dims) // keep in mind that num centroids*dims has to be lest than num_clusters
    {
        centroids[idx] /= counts[idx/dims];
        if (idx < num_centroids)
        {
            counts[idx] = 0;
        }
    }
    assigned[idx] = 0;
}

int main()
{
    const int N_CLUSTERS = 3;
    const int N_ITERATIONS = 100;
    const std::string path = "iris_edit.csv";
    std::tuple<int, int> pts_dims = getCSVDimensions(path);
    const int N_DATAPTS = std::get<0>(pts_dims);
    const int DIMS = std::get<1>(pts_dims);
    std::vector<double> datapoints = readCSVTo1DArray(path);
    std::vector<int> indices = fisherYatesShuffle(N_DATAPTS, N_CLUSTERS);
    std::vector<double> centroids;

    for (int i = 0; i < N_CLUSTERS; i++)
    {
        for (int j = 0; j < DIMS; j++)
        {
            centroids.push_back(datapoints[indices[i] + j]);
        }
    }

    double* dataptsGPU = 0;
    double* centroidsGPU = 0;
    int* assignedGPU = 0;
    int* countsGPU = 0;

    hipMalloc(&dataptsGPU, N_DATAPTS * DIMS * sizeof(double));
    hipMalloc(&centroidsGPU, N_CLUSTERS * DIMS * sizeof(double));
    hipMalloc(&assignedGPU, N_DATAPTS * sizeof(int));
    hipMalloc(&countsGPU, N_CLUSTERS * sizeof(int));

    hipMemcpy(dataptsGPU, datapoints.data(), N_DATAPTS * DIMS * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(centroidsGPU, centroids.data(), N_CLUSTERS * DIMS * sizeof(double), hipMemcpyHostToDevice);

    for (int i = 0; i < N_ITERATIONS; i++)
    {
        update <<<1, N_DATAPTS>>> (dataptsGPU, centroidsGPU, assignedGPU, countsGPU, N_DATAPTS, N_CLUSTERS, DIMS);
    }

    return 0;
}